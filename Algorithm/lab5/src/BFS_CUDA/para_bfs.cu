#include "hip/hip_runtime.h"
#include<stdlib.h> 
#include<stdio.h> 
#include<iostream>
#include<string> 
#include<math.h>
#include<fstream>
#include<sstream>
#include<hip/hip_runtime.h>
#include<time.h>

#define SIZE 82000

using namespace std;

size_t threadsPerBlock;
size_t numberOfBlocks;

enum color {
    white,
    black,
    grew
};

typedef struct edge_node
{
    int vex;
    edge_node *next;
} edge_node;

typedef struct vex_node
{
    int vex_num;
    color col;
    edge_node *edges;
} vex_node; 


int graph_size;

__global__ void search_kernel(int N,vex_node *g,bool &findx,bool &findy,int &tmpx,int &tmpy)
{
    int index= threadIdx.x + blockIdx.x * blockDim.x;
    int gridStride = gridDim.x * blockDim.x;
    for (int i = index; i < N; i += gridStride)
    {
        if(g[i].vex_num==y)
        {
            findy=true;
            tmpy=i;
        }
        if(g[i].vex_num==x)
        {
            findx=true;
            tmpx=i;
        }
        //if(findx&&findy) break;
    }
}
void creategraph(char *filename,vex_node *g)
{
    int i;
    int x,y;
    int sta=0;
    int tmpx,tmpy;
    bool findx,findy;
    ifstream in(filename);
    
    while(!in.eof())
    {
        // new input
        in>>x>>y;
        // map old point
        findx=false;
        findy=false;

        search_kernel<<<numberOfBlocks, threadsPerBlock>>>(sta,g,findx,findy,tmpx,tmpy);
        /*for(i=0;i<sta;i++)
        {
            if(g[i].vex_num==y)
            {
                findy=true;
                tmpy=i;
            }
            if(g[i].vex_num==x)
            {
                findx=true;
                tmpx=i;
            }
            if(findx&&findy) break;
        }*/ 

        //add new node
        if(!findx) 
        {
            tmpx=sta;
            g[tmpx].vex_num=x;
            g[tmpx].col=white;
            g[tmpx].edges=NULL;
            sta++;
        } 
        if(!findy)
        {
            tmpy=sta;
            g[tmpy].vex_num=y;
            g[tmpy].col=white;
            g[tmpy].edges=NULL;
            sta++;
        }

        // update node edges table 
        edge_node *p=g[tmpx].edges;
        edge_node *q;
        hipMallocManaged(&q, sizeof(edge_node));
        q->vex=tmpy;
        q->next=NULL;
        if(p==NULL) g[tmpx].edges=q;
        else
        {
            while(p->next!=NULL)
                p=p->next;
            p->next=q;
        }
        
        // cheak map
        if(y!=g[tmpy].vex_num) 
            printf("Error:map wrong!\n");

        //printf("current vex number: %d\n",sta);
    }
    graph_size=sta;
}

void printgraph(vex_node *g)
{
    int i;
    int total=0,max=0;
    for(i=0;i<graph_size;i++)
    {
        //printf("No.%d: num=%d,edges:",i,g[i].vex_num);
        edge_node *p=g[i].edges;
        total=0;
        while(p!=NULL)
        {
            //printf("%d ",g[p->vex].vex_num);
            total++;
            p=p->next;
        }
        if(max<total) max=total;
        //printf("\n");
    }
    printf("max deepth:%d",max);
}

void bfs(int src)
{

}

int main(int argc,char *argv[])
{
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;

    clock_t begin,end;
    double time;

    vex_node *g;
    size_t size = SIZE * sizeof(vex_node);
    hipMallocManaged(&g, size);

    begin=clock();
    creategraph(argv[1],g);
    end=clock();
    time=(double)(end-begin)/CLOCKS_PER_SEC;
    printf("\nread time:%.8fs",time);

    printgraph(g);
}

