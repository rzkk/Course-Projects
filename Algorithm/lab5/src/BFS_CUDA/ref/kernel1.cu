#include "hip/hip_runtime.h"
#ifndef _KERNEL1_H_
#define _KERNEL1_H_

__global__ void Kernel1(Node* d_node,int* d_edges,bool* d_frontier, bool* d_visited,int* d_cost,bool* d_stop)
{
	int tid=threadIdx.x; //�ҵ�ͼ4����㣬��Ӧһ��block���4���̡߳�
	if(d_frontier[tid]==true && d_visited[tid]==false)
	{
		d_frontier[tid]=false;
		for(int i=d_node[tid].start;i<d_node[tid].start+d_node[tid].num;i++)
		{
			int id=d_edges[i];
			if(d_frontier[id]==false && d_visited[id]==false)
			{
				d_cost[id]=d_cost[tid]+1;
				d_frontier[id]=true;
				*d_stop=true;

			}
		}
		d_visited[tid]=true;
		
	}

}

#endif