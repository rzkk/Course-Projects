/* 
Author: Juliet(������)
QQ: 773917068
Date: Mar 10
About: BFS_CUDA

*/
#include <stdio.h>
#include <cutil.h>


struct Node
{
	int start;
	int num;
};
#include <kernel1.cu>
void BFSGraph(int argc, char** argv);

int main(int argc, char** argv)
{
	BFSGraph(argc,argv);
	CUT_EXIT(argc,argv);
}

void BFSGraph(int argc, char** argv)
{

	//the graph description
	struct Node node[4];
	node[0].start=0;
	node[0].num=2;
	node[1].start=2;
	node[1].num=1;
	node[2].start=3;
	node[2].num=1;
	node[3].start=3; //attention node[i]'s judgement
	node[3].num=0;
	int edges[]={1,2,3,3};

	bool frontier[4]={false};
	bool visited[4]={false};
	int cost[4]={0};

	int source=0;
	frontier[source]=true;
	
	Node* d_node;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_node,sizeof(Node)*4));
	CUDA_SAFE_CALL(hipMemcpy(d_node,node,sizeof(Node)*4,hipMemcpyHostToDevice));

	int* d_edges;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_edges,sizeof(Node)*4));
	CUDA_SAFE_CALL(hipMemcpy(d_edges,edges,sizeof(Node)*4,hipMemcpyHostToDevice));

	bool* d_frontier;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_frontier,sizeof(bool)*4));
	CUDA_SAFE_CALL(hipMemcpy(d_frontier,frontier,sizeof(bool)*4,hipMemcpyHostToDevice));

	bool* d_visited;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_visited,sizeof(bool)*4));
	CUDA_SAFE_CALL(hipMemcpy(d_visited,visited,sizeof(bool)*4,hipMemcpyHostToDevice));

	int* d_cost;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_cost,sizeof(int)*4));
	CUDA_SAFE_CALL(hipMemcpy(d_cost,cost,sizeof(int)*4,hipMemcpyHostToDevice));

	dim3 grid(1,1,1);
	dim3 threads(4,1,1);

	bool stop;
	bool* d_stop;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_stop,sizeof(bool)));

	do{
		stop=false;
		CUDA_SAFE_CALL(hipMemcpy(d_stop,&stop,sizeof(bool),hipMemcpyHostToDevice));
		Kernel1<<<grid,threads,0>>>(d_node,d_edges,d_frontier,d_visited,d_cost,d_stop);
		CUT_CHECK_ERROR("kernel1 execution failed");


		CUDA_SAFE_CALL(hipMemcpy(&stop,d_stop,sizeof(bool),hipMemcpyDeviceToHost));
	}while(stop);

	CUDA_SAFE_CALL(hipMemcpy(cost,d_cost,sizeof(int)*4,hipMemcpyDeviceToHost));

	for(int i=0;i<4;i++)
		printf("%d	",cost[i]);

}