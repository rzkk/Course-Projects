#include "hip/hip_runtime.h"
#include<stdlib.h> 
#include<stdio.h> 
#include<string.h> 
#include<math.h> 
#include<cutil.h> 
#include<hip/hip_runtime.h>
#include<time.h>

struct node
{
    int start;
    int num_edges;
}; 

__global__ 
void Kernel(node *g_node,int *g_edges,bool *g_frontier,bool *g_visited,int *g_cost,bool *g_over,int num_nodes)
{
    int tid=blockIdx.x*128+threadIdx.x;
    if(tid<num_nodes&&g_frontier[tid])
    {
        g_frontier[tid]=false;
        g_visited[tid]=true;
        for(int i=g_node[tid].start;i<g_node[tid].start+g_node[tid].num_edges;i++)
        {
            int id=g_edges[i];
            if(!g_visited[id])
            {
                g_cost[id]=g_cost[tid]+1;
                g_frontier[id]=true;
                *g_over=true;
            }
        }
    }
}
void bfs_graph(int num_vertex,int num_edges_list)
{
    int i,k,vertex,edges,id,cost,source;
    scanf("%d",&num_vertex);
    node *h_node=(node*)malloc(sizeof(node)*num_vertex);
    bool *h_frontier=(bool*)malloc(sizeof(bool)*num_vertex);
    bool *h_visited=(bool*)malloc(sizeof(bool)*num_vertex);
    int *h_cost = (int*)malloc( sizeof(int)*num_vertex);
    for(i=0;i<num_vertex;i++)
    {
        scanf("%d %d",&vertex,&edges);
        h_node[i].start=vertex;
        h_node[i].num_edges=edges;
        h_frontier[i]=false;
        h_visited[i]=false;
        h_cost[i]=-1;
    }
    scanf("%d",&source);
    h_frontier[source]=true;
    h_cost[source]=0;
    scanf("%d",&num_edges_list);
    int* h_edges=(int*)malloc(sizeof(int)*num_edges_list);
    for(i=0;i<num_edges_list;i++)
    {
        scanf("%d %d",&id,&cost);
        h_edges[i]=id;
    }
    clock_t begin,end;
    double time;
    begin=clock();
    node *d_node;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_node,sizeof(node)*num_vertex));
    CUDA_SAFE_CALL(hipMemcpy(d_node,h_node,sizeof(node)*num_vertex,hipMemcpyHostToDevice));
    int *d_edges,*d_cost;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_edges,sizeof(int)*num_edges_list));
    CUDA_SAFE_CALL(hipMemcpy(d_edges,h_edges,sizeof(int)*num_edges_list,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_cost,sizeof(int)*num_vertex));
    CUDA_SAFE_CALL(hipMemcpy(d_cost,h_cost,sizeof(int)*num_vertex,hipMemcpyHostToDevice));
    bool *d_frontier,*d_visited;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_frontier,sizeof(bool)*num_vertex));
    CUDA_SAFE_CALL(hipMemcpy(d_frontier,h_frontier,sizeof(bool)*num_vertex,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_visited,sizeof(bool)*num_vertex));
    CUDA_SAFE_CALL(hipMemcpy(d_visited,h_visited,sizeof(bool)*num_vertex,hipMemcpyHostToDevice));
    bool *d_over;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_over,sizeof(bool)));
    dim3 grid(128,1,1);
    dim3 threads(128,1,1);
    bool stop;
    do
    {
        stop=false;
        hipMemcpy(d_over,&stop,sizeof(bool),hipMemcpyHostToDevice); 
        Kernel<<<grid,threads,0>>>(d_node,d_edges,d_frontier,d_visited,d_cost,d_over,num_vertex);
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        CUT_CHECK_ERROR("Kernel execution failed");
        CUDA_SAFE_CALL(hipMemcpy(&stop,d_over,sizeof(bool),hipMemcpyDeviceToHost));
        k++;
    } while(stop);
    CUDA_SAFE_CALL(hipMemcpy(h_cost,d_cost,sizeof(int)*num_vertex,hipMemcpyDeviceToHost));
    for(i=0;i<num_vertex;i++)
        printf("%d cost:%d\n",i,h_cost[i]);
    end=clock();
    time=(double)(end-begin)/CLOCKS_PER_SEC;
    printf("\n%f",time);
}

int main()
{
    int num_vertex=0;
    int num_edges_list=0;
    bfs_graph(num_vertex,num_edges_list);
    return 0;
}