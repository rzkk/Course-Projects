#include<stdlib.h> 
#include<stdio.h> 
#include<iostream>
#include<string> 
#include<math.h>
#include<fstream>
#include<sstream>
#include<hip/hip_runtime.h>
#include<time.h>

#define SIZE 82000

using namespace std;

size_t threadsPerBlock;
size_t numberOfBlocks;
int deviceId;

enum color {
    white,
    black,
    grew
};

typedef struct edge_node
{
    int vex;
    edge_node *next;
} edge_node;

typedef struct vex_node
{
    int vex_num;
    color col;
    edge_node *edges;
    edge_node *tail;
} vex_node; 


int graph_size;

__global__ void search_kernel(int N,vex_node *g,bool *findx,bool *findy,int *tmpx,int *tmpy,int x,int y)
{
    int index= threadIdx.x + blockIdx.x * blockDim.x;
    int gridStride = gridDim.x * blockDim.x;
    for (int i = index; i < N; i += gridStride)
    {
        if(g[i].vex_num==y)
        {
            *findy=true;
            *tmpy=i;
        }
        if(g[i].vex_num==x)
        {
            *findx=true;
            *tmpx=i;
        }
        //if(findx&&findy) break;
    }
}

void printgraph(vex_node *g)
{
    int i;
    int total=0,max=0;
    for(i=0;i<graph_size;i++)
    {
        printf("No.%d: num=%d,edges:",i,g[i].vex_num);
        edge_node *p=g[i].edges;
        total=0;
        while(p!=NULL)
        {
            printf("%d ",g[p->vex].vex_num);
            total++;
            p=p->next;
        }
        if(max<total) max=total;
        printf("\n");
    }
}

void creategraph(char *filename,vex_node *g)
{
    int i;
    int x=1,y;
    int sta;
    //int tmpx=0,tmpy=0;
    //bool findx,findy;

    //int *d_x,*d_y;
    int *d_tmpx,*d_tmpy;
    bool *d_findx,*d_findy;
    //cudaMallocManaged(&d_x,sizeof(int));
    //cudaMallocManaged(&d_y,sizeof(int));
    hipMallocManaged(&d_tmpx,sizeof(int));
    hipMallocManaged(&d_tmpy,sizeof(int));
    hipMallocManaged(&d_findx,sizeof(bool));
    hipMallocManaged(&d_findy,sizeof(bool));
    *d_tmpx=0;
    *d_tmpy=0;

    sta=0;
    edge_node *q;
    ifstream in(filename);
    //printf("here2\n");
    while(!in.eof())
    {
        // new input
        in>>x>>y;
        // map old point
        *d_findx=false;
        *d_findy=false;

        /*cudaMemcpy(d_x,&x,sizeof(int),cudaMemcpyHostToDevice);
        cudaMemcpy(d_y,&y,sizeof(int),cudaMemcpyHostToDevice);
        cudaMemcpy(d_findx,&findx,sizeof(bool),cudaMemcpyHostToDevice);
        cudaMemcpy(d_findy,&findy,sizeof(bool),cudaMemcpyHostToDevice);*/

        numberOfBlocks=sta/threadsPerBlock+1;
        search_kernel<<<numberOfBlocks, threadsPerBlock>>>(sta,g,d_findx,d_findy,d_tmpx,d_tmpy,x,y);
        hipDeviceSynchronize();

        /*cudaMemcpy(&tmpx,d_tmpx,sizeof(int),cudaMemcpyDeviceToHost);
        cudaMemcpy(&tmpy,d_tmpy,sizeof(int),cudaMemcpyDeviceToHost);
        cudaMemcpy(&findx,d_findx,sizeof(bool),cudaMemcpyDeviceToHost);
        cudaMemcpy(&findy,d_findy,sizeof(bool),cudaMemcpyDeviceToHost);*/
        
        //add new node
        if(!*d_findx) 
        {
            *d_tmpx=sta;
            g[*d_tmpx].vex_num=x;
            g[*d_tmpx].col=white;
            g[*d_tmpx].edges=NULL;
            g[*d_tmpx].tail=NULL;
            sta++;
        } 
        //printf("here4\n");
        if(!*d_findy)
        {
            *d_tmpy=sta;
            g[*d_tmpy].vex_num=y;
            g[*d_tmpy].col=white;
            g[*d_tmpy].edges=NULL;
            g[*d_tmpx].tail=NULL;
            sta++;
        }
        
        // update node edges table 
        edge_node *p=g[*d_tmpx].tail;
        hipMallocManaged(&q, sizeof(edge_node));
        q->vex=*d_tmpy;
        q->next=NULL;

        if(p==NULL) 
        {
            g[*d_tmpx].edges=q;
            g[*d_tmpx].tail=q;
        }
        else
        {
            p->next=q;
            g[*d_tmpx].tail=q;
        }
        
        // cheak map
        if(y!=g[*d_tmpy].vex_num) 
            printf("Error:map wrong!\n");
        //graph_size=*sta;
        //printgraph(g);
        printf("current vex number: %d\n",sta);
    }
    graph_size=sta;
}

__global__ void bfs_kernel(vex_node *g,int N,bool *change)
{
    int index= threadIdx.x + blockIdx.x * blockDim.x;
    int gridStride = gridDim.x * blockDim.x;
    for (int i = index; i < N; i += gridStride)
    {
        if(g[i].col==grew)
        {
            edge_node *p=g[i].edges;
            while(p!=NULL)
            {
                if(g[p->vex].col==white) {
                    *change=true;
                    //printf("get point\n");
                }
                g[p->vex].col=grew;
                p=p->next;
            }
            g[i].col==black;
        }
    }
}

void bfs(vex_node *g)
{
    int i;
    bool *change;
    int total=0;
    hipMallocManaged(&change,sizeof(bool));
    for(i=0;i<graph_size;i++)
    {
        if(g[i].col==white)
        {
            g[i].col=grew;
            *change=true;
            while(*change)
            {
                *change=false;
                //if(!*change) printf("clear\n");
                total++;
                hipDeviceSynchronize();
                bfs_kernel<<<numberOfBlocks, threadsPerBlock>>>(g,graph_size,change);
                hipDeviceSynchronize();
                //if(*change) printf("next bfs\n");
            }
        }
    }
    printf("\ntotal bfs times:%d\n",total);
}

int main(int argc,char *argv[])
{
    
    int numberOfSMs;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;

    clock_t begin,end;
    double time1,time2;

    vex_node *g;
    size_t size = SIZE * sizeof(vex_node);
    hipMallocManaged(&g, size);

    begin=clock();
    creategraph(argv[1],g);
    end=clock();
    time1=(double)(end-begin)/CLOCKS_PER_SEC;

    printgraph(g);

    begin=clock();
    bfs(g);
    end=clock();
    time2=(double)(end-begin)/CLOCKS_PER_SEC;
    printf("\nread time:%.8fs",time1);
    printf("\nbfs time:%.8fs\n",time2);
}

